// -*- c++ -*-
/*

Strategy is to operate on a single voltage file, and produce a heap of stuff. 
 - read in and send to GPU
 - simple promote
 - optionally calibrate voltages
 - correlate and write out
 - optionally remove delays from visibilities
 - optionally average visibilities in frequency
 - rotate visibilities to particular beam (later can be RA/DEC)
 - write out beamformed filterbank

*/

#include <iostream>
#include <algorithm>
using std::cout;
using std::cerr;
using std::endl;
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <string.h>
#include <unistd.h>
#include <netdb.h>
#include <sys/socket.h>
#include <sys/types.h>
#include <netinet/in.h>
#include <time.h>
#include <syslog.h>
#include <pthread.h>

#include <mma.h>
#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"

#include <hip/hip_runtime_api.h>
using namespace nvcuda;

#define NANT 63
#define NCHAN 384
#define NT 30720
#define NBASE 2016
#define NPTR 8 // pols, small times, r/i
#define sep 1.0 // arcmin
#define AV 8
#define PI 3.141592653589793238
#define CVAC 299792458.0


// dedisperser
// dms are integer shifts
// run with NT*NANT blocks of NCHAN threads
__global__ void dedisperser(char *input, char *output, int *dms) {

  size_t bidx = blockIdx.x; 
  size_t ch = threadIdx.x;
  size_t iidx = bidx*NCHAN+ch;

  // time sample
  size_t tim = (size_t)(bidx / NANT);
  // antenna
  size_t ant = (size_t)(bidx % NANT);
  size_t oidx;

  // wrap
  if (tim < dms[ch]) {
    oidx = (NT-(dms[ch]-tim))*NANT*NCHAN + ant*NCHAN + ch;
    for (size_t i=0;i<4;i++) 
      output[4*oidx+i] = input[4*iidx+i];
  }

  // normal shift
  if (tim >= dms[ch]) {
    oidx = (tim-dms[ch])*NANT*NCHAN + ant*NCHAN + ch;
    for (size_t i=0;i<4;i++) 
      output[4*oidx+i] = input[4*iidx+i];
  }
 
}


// promoter to fp32
// run with NANT*NCHAN*NPTR/2/32 blocks of 32 threads
__global__ void promoter(char *input, float *output) {

  int bidx = blockIdx.x; // assume 16*48*NANT
  int tidx = threadIdx.x; // assume 32
  int iidx = bidx*32+tidx;
  
  output[2*iidx] = (float)((char)(((unsigned char)(input[iidx]) & (unsigned char)(15)) << 4) >> 4); //r
  output[2*iidx+1] = (float)((char)(((unsigned char)(input[iidx]) & (unsigned char)(240))) >> 4); //i

}

// correlator
// input is two packed time ints for all antennas
// also input antenna 1 and 2 indices for each baseline
// output is [2x time, baseline, freq, pols, r/i]
// run with NBASE*NCHAN/32 blocks of 32 threads
__global__ void correlator(float *input, float *output, int *a1, int *a2, float scfac, float *weights) {

  int bidx = blockIdx.x; // assume 16*48*NANT                                                              
  int tidx = threadIdx.x; // assume 32                                                                     
  int iidx = bidx*32+tidx;
  int basel = (int)(iidx/NCHAN); // baseline number
  int chgidx = (int)(bidx % (NCHAN/32)); // index of 32-channel group for this block
  int ch = (int)(iidx % NCHAN); // channel number
  
  // each block operates on 32 channels (one per thread)
  __shared__ float d1[32*NPTR];
  __shared__ float d2[32*NPTR];
  // start indices for each antenna from input
  int idx0_1 = a1[basel]*NCHAN*NPTR + chgidx*32*NPTR;
  int idx0_2 = a2[basel]*NCHAN*NPTR + chgidx*32*NPTR;

  // pull data into shared mem, for each antenna
  int ii = tidx*NPTR;
  for (int i=idx0_1+tidx*NPTR; i<idx0_1+(tidx+1)*NPTR; i++) {
    d1[ii] = input[i];
    ii++;
  }
  ii=tidx*NPTR;
  for (int i=idx0_2+tidx*NPTR; i<idx0_2+(tidx+1)*NPTR; i++) {
    d2[ii] = input[i];
    ii++;
  }


  // get weights for a1 and a2;
  float w_a1[4], w_a2[4];
  for (int i=0;i<4;i++) {
    w_a1[i] = weights[a1[basel]*192 + (int)(ch/8)*4 + i];
    w_a2[i] = weights[a2[basel]*192 + (int)(ch/8)*4 + i];
  }
  
  // now each thread can happily operate on a single channel
  // order is [time, pol, R/I]
  // make two separate arrays, each with [X*X / X*Y / Y*X / Y*Y, complexity]
  float output_tims[2][8], a1r, a1i, a2r, a2i;
  float w1r, w1i, w2r, w2i;
  // loop over times
  for (int ti=0;ti<2;ti++) {
    // loop over pols
    ii=0;
    for (int p1=0;p1<2;p1++) {
      for (int p2=0;p2<2;p2++) {
	
	a1r = d1[tidx*NPTR + ti*4 + p1*2];
	a1i = d1[tidx*NPTR + ti*4 + p1*2 + 1];
	a2r = d2[tidx*NPTR + ti*4 + p2*2];
	a2i = d2[tidx*NPTR + ti*4 + p2*2 + 1];

	w1r = a1r*w_a1[2*p1] - a1i*w_a1[2*p1+1];
	w2r = a2r*w_a2[2*p2] - a2i*w_a2[2*p2+1]; 
	w1i = a1r*w_a1[2*p1+1] + a1i*w_a1[2*p1];
	w2i = a2r*w_a2[2*p2+1] + a2i*w_a2[2*p2]; 
	/*w1r = a1r;
	w1i = a1i;
	w2r = a2r;
	w2i = a2i;*/
	
	output_tims[ti][2*ii] = w1r*w2r + w1i*w2i;
	output_tims[ti][2*ii+1] = w1r*w2i - w1i*w2r;

	ii++;
	
      }
    }
  }
  
  // write to output
  ii = basel*NCHAN*8 + ch*8;
  for (int i=0;i<8;i++) output[ii+i] += output_tims[0][i]*scfac;
  ii += NBASE*NCHAN*8;
  for (int i=0;i<8;i++) output[ii+i] += output_tims[1][i]*scfac;
  
  
}

// input has shape NBASE*NCHAN*8
// reduce to stokes I along NBASE axis using shared memory
// run with NCHAN blocks of 512 threads - will add 2016 baselines
__global__ void reduce_corrs(float *input, float *output, float scfac, int *a1, int *a2, int stokes, float *antpos, float minBase) {

  int bidx = blockIdx.x; // assume NCHAN
  int tidx = threadIdx.x; // assume 512                                                                  
  int iidx = bidx*512+tidx;

  volatile __shared__ float summer[512];

  // add into shared memory
  summer[tidx] = 0.;

  // stokes I
  if (stokes==0) {
    if (tidx<504) {
      if (a1[tidx]!=a2[tidx] && fabsf(antpos[a2[tidx]]-antpos[a1[tidx]])>minBase)
	summer[tidx] += input[tidx*NCHAN*8 + bidx*8] + input[tidx*NCHAN*8 + bidx*8 + 6];
      if (a1[tidx+504]!=a2[tidx+504] && fabsf(antpos[a2[tidx+504]]-antpos[a1[tidx+504]])>minBase)
	summer[tidx] += input[(tidx+1*504)*NCHAN*8 + bidx*8] + input[(tidx+1*504)*NCHAN*8 + bidx*8 + 6];
      if (a1[tidx+2*504]!=a2[tidx+2*504] && fabsf(antpos[a2[tidx+2*504]]-antpos[a1[tidx+2*504]])>minBase)
	summer[tidx] += input[(tidx+2*504)*NCHAN*8 + bidx*8] + input[(tidx+2*504)*NCHAN*8 + bidx*8 + 6];
      if (a1[tidx+3*504]!=a2[tidx+3*504] && fabsf(antpos[a2[tidx+3*504]]-antpos[a1[tidx+3*504]])>minBase)
	summer[tidx] += input[(tidx+3*504)*NCHAN*8 + bidx*8] + input[(tidx+3*504)*NCHAN*8 + bidx*8 + 6];
    }
  }
  // stokes Q
  if (stokes==1) {
    if (tidx<504) {
      if (a1[tidx]!=a2[tidx])
	summer[tidx] += input[tidx*NCHAN*8 + bidx*8] - input[tidx*NCHAN*8 + bidx*8 + 6];
      if (a1[tidx+504]!=a2[tidx+504])
	summer[tidx] += input[(tidx+1*504)*NCHAN*8 + bidx*8] - input[(tidx+1*504)*NCHAN*8 + bidx*8 + 6];
      if (a1[tidx+2*504]!=a2[tidx+2*504])
	summer[tidx] += input[(tidx+2*504)*NCHAN*8 + bidx*8] - input[(tidx+2*504)*NCHAN*8 + bidx*8 + 6];
      if (a1[tidx+3*504]!=a2[tidx+3*504])
	summer[tidx] += input[(tidx+3*504)*NCHAN*8 + bidx*8] - input[(tidx+3*504)*NCHAN*8 + bidx*8 + 6];
    }
  }
  // stokes U
  if (stokes==2) {
    if (tidx<504) {
      if (a1[tidx]!=a2[tidx])
	summer[tidx] += input[tidx*NCHAN*8 + bidx*8 + 2] + input[tidx*NCHAN*8 + bidx*8 + 4];
      if (a1[tidx+504]!=a2[tidx+504])
	summer[tidx] += input[(tidx+1*504)*NCHAN*8 + bidx*8 + 2] + input[(tidx+1*504)*NCHAN*8 + bidx*8 + 4];
      if (a1[tidx+2*504]!=a2[tidx+2*504])
	summer[tidx] += input[(tidx+2*504)*NCHAN*8 + bidx*8 + 2] + input[(tidx+2*504)*NCHAN*8 + bidx*8 + 4];
      if (a1[tidx+3*504]!=a2[tidx+3*504])
	summer[tidx] += input[(tidx+3*504)*NCHAN*8 + bidx*8 + 2] + input[(tidx+3*504)*NCHAN*8 + bidx*8 + 4];
    }
  }
  // stokes V
  if (stokes==3) {
    if (tidx<504) {
      if (a1[tidx]!=a2[tidx])
	summer[tidx] += input[tidx*NCHAN*8 + bidx*8 + 3] - input[tidx*NCHAN*8 + bidx*8 + 5];
      if (a1[tidx+504]!=a2[tidx+504])
	summer[tidx] += input[(tidx+1*504)*NCHAN*8 + bidx*8 + 3] - input[(tidx+1*504)*NCHAN*8 + bidx*8 + 5];
      if (a1[tidx+2*504]!=a2[tidx+2*504])
	summer[tidx] += input[(tidx+2*504)*NCHAN*8 + bidx*8 + 3] - input[(tidx+2*504)*NCHAN*8 + bidx*8 + 5];
      if (a1[tidx+3*504]!=a2[tidx+3*504])
	summer[tidx] += input[(tidx+3*504)*NCHAN*8 + bidx*8 + 3] - input[(tidx+3*504)*NCHAN*8 + bidx*8 + 5];
    }
  }

  // [X_i X_j*  X_i Y_j*  Y_i X_j* Y_i Y_j*]
  // Stokes I: 0.5*(Re(X_i X_j*) + Re(Y_i Y_j*))
  // Stokes Q: 0.5*(Re(X_i X_j*) - Re(Y_i Y_j*))
  // Stokes U: 0.5*(Re(X_i Y_j*) + Re(Y_i X_j*))
  // Stokes V: 0.5*(Im(X_i Y_j*) - Im(Y_i X_j*))
  
  __syncthreads();

  // now reduce in shared memory
  if (tidx<256) {
    summer[tidx] += summer[tidx+256];
    __syncthreads();
    summer[tidx] += summer[tidx+128];
    __syncthreads();
    summer[tidx] += summer[tidx+64];
    __syncthreads();
    summer[tidx] += summer[tidx+32];
    __syncthreads();
    summer[tidx] += summer[tidx+16];
    __syncthreads();
    summer[tidx] += summer[tidx+8];
    __syncthreads();
    summer[tidx] += summer[tidx+4];
    __syncthreads();
    summer[tidx] += summer[tidx+2];
    __syncthreads();
    summer[tidx] += summer[tidx+1];
  }

  __syncthreads();

  if (tidx==0) output[bidx] = (summer[0]*scfac);

}

// this kernel removes baseline delays by multiplying by exp(-2*pi*i*nu*tau)
// run with NBASE*NCHAN*4/32 blocks of 32 threads
// delays in ns
__global__ void delayer(float *input, float *freqs, float *delays) {

  int bidx = blockIdx.x; // assume 16*48*NANT
  int tidx = threadIdx.x; // assume 32
  int iidx = bidx*32+tidx;
  int bci = (int)(iidx/4);
  int basel = (int)(bci / NCHAN);
  int ch = (int)(bci % NCHAN);

  float vr, vi, arg=-2.*3.14159265359*freqs[ch]*delays[basel]*1e-9;
  vr = input[2*iidx]*cosf(arg) - input[2*iidx+1]*sinf(arg);
  __syncthreads();
  vi = input[2*iidx]*sinf(arg) + input[2*iidx+1]*cosf(arg);
  __syncthreads();

  input[2*iidx] = vr;
  input[2*iidx+1] = vi;
  
}

// kernel to enable frequency averaging of visibility output
// will only output XX and YY pols
// run with NBASE*NCHAN*4/AV/32 blocks of 32 threads
__global__ void fscrunch(float *input, float *output) {

  int bidx = blockIdx.x; 
  int tidx = threadIdx.x; // assume 32
  int iidx = bidx*32+tidx;
  int bcli = (int)(iidx/4);
  int poli = (int)(iidx % 4);
  int basel = (int)(bcli / (NCHAN/AV));
  int lch = (int)(bcli % (NCHAN/AV));

  int sumss[4];
  sumss[0] = 0;
  sumss[1] = 1;
  sumss[2] = 6;
  sumss[3] = 7;

  output[iidx] = 0.;
  for (int i=0;i<AV;i++) 
    output[iidx] += input[basel*NCHAN*8 + (AV*lch+i)*8 + sumss[poli]];
    
}

// really simple - adds the two times in correlator output
// run with NBASE*NCHAN*8/32 blocks of 32 threads
__global__ void adder(float *input, float *output) {

  int bidx = blockIdx.x; // assume 16*48*NANT
  int tidx = threadIdx.x; // assume 32
  int iidx = bidx*32+tidx;
  
  output[iidx] = input[iidx] + input[NBASE*NCHAN*8 + iidx];

}

// really simple - zeros correlator output
// run with 2*NBASE*NCHAN*8/32 blocks of 32 threads
__global__ void zeroer(float *input) {

  int bidx = blockIdx.x; // assume 16*48*NANT                                                              
  int tidx = threadIdx.x; // assume 32                                                                     
  int iidx = bidx*32+tidx;

  input[iidx] = 0.;

}


// CPU functions
int init_weights(char *wnam, float *antpos, float *weights, char *flagnam, int weight, int doflag, int donorm);
// loads in weights
int init_weights(char * wnam, float *antpos, float *weights, char *flagnam, int weight, int doflag, int donorm) {

  // assumes 64 antennas
  // antpos: takes only easting
  // weights: takes [ant, NW==48] 

  FILE *fin;
  FILE *fants;
  float wnorm;

  if (weight) {
    if (!(fin=fopen(wnam,"rb"))) {
      printf("Couldn't open weights file %s\n",wnam);
      return 1;
    }

    fread(antpos,64*sizeof(float),1,fin);
    fread(weights,64*48*2*2*sizeof(float),1,fin);

    for (int i=0;i<64*48*2;i++) {
      wnorm = sqrt(weights[2*i]*weights[2*i] + weights[2*i+1]*weights[2*i+1]);
      if (wnorm!=0.0 && donorm==1) {
	weights[2*i] /= wnorm*wnorm;
	weights[2*i+1] /= wnorm*wnorm;
      }
    }           

    fclose(fin);
  }
  else {

    for (int i=0;i<64*48*2;i++) {
      weights[2*i] = 1.;
      weights[2*i+1] = 0.;
    }

    for (int i=0;i<64;i++) {
      antpos[i] = 0.;
    }

  }
 

  int ant;
  if (doflag) {
    if (!(fants=fopen(flagnam,"r"))) {
      printf("Couldn't open flag ants file %s\n",flagnam);
      return 1;
    }
    
    while (!feof(fants)) {
      fscanf(fants,"%d\n",&ant);
      for (int j=0;j<48*2*2;j++) {
	weights[ant*48*2*2+j] = 0.0;
      }
    }

    fclose(fants);
    
  }

  //for (int i=0;i<63;i++) 
  //  printf("%f\n",antpos[i]);
  
  printf("Loaded antenna positions and weights\n");
  return 0;

}

void calc_voltage_weights(float *antpos, float *weights, float *freqs, float *bfweights, int nBeamNum);
void calc_voltage_weights(float *antpos, float *weights, float *freqs, float *bfweights, int nBeamNum) {

  float theta, afac, twr, twi;
  theta = sep*(127.-(float)nBeamNum)*3.14159265358/10800.; // radians
  for(int nAnt=0;nAnt<64;nAnt++){
    for(int nChan=0;nChan<48;nChan++){
      for(int nPol=0;nPol<2;nPol++){
	afac = -2.*3.14159265358*freqs[nChan*8+4]*theta/CVAC; // factor for rotate
	twr = cos(afac*antpos[nAnt]);
	twi = sin(afac*antpos[nAnt]);

	bfweights[nAnt*(48*2*2)+nChan*2*2+nPol*2] = (twr*weights[(nAnt*(48*2)+nChan*2+nPol)*2] - twi*weights[(nAnt*(48*2)+nChan*2+nPol)*2+1]);
	bfweights[nAnt*(48*2*2)+nChan*2*2+nPol*2+1] = (twi*weights[(nAnt*(48*2)+nChan*2+nPol)*2] + twr*weights[(nAnt*(48*2)+nChan*2+nPol)*2+1]);
      }
    }
  }

}

// only does Stokes I for now
void sum_to_filterbank(float *corrout, unsigned char *filout) {

  float val;
  for (int iCh=0; iCh<NCHAN; iCh++) {
    val = 0.;
    for (int b=0; b<NBASE; b++) 
      val += (0.5*(corrout[b*NCHAN*8 + iCh*8] + corrout[b*NCHAN*8 + iCh*8 + 6]));
    filout[iCh]	= (unsigned char)(val);
  }

}

void usage()
{
  fprintf (stdout,
	   "toolkit [options]\n"
	   " -i input filename [no default]\n"
	   " -o output filename [no default - will not write if none given]\n"
	   " -t number of time integrations x 32.768us [default 8]\n"
	   " -w optional weights file\n"
	   " -f optional antenna flags file\n"
	   " -c set frequency of first channel in MHz [default 1530.0]\n"
	   " -b optionally set in 0-255 to rotate voltages to beam\n"
	   " -p coherent philterbank writing file [no default - will not write if none given]\n"
	   " -d file with NBASE delays to remove from baselines [optional - no default]\n"
	   " -a average visibilities by 8x in frequency\n"
	   " -m dedisperse [optional - no default]\n"
	   " -u unify all antennas (testing only)\n"
	   " -s output number of packets to be processed. 1 packet = 2 samples [default NT==30720]\n"
	   " -q offset from start in number of packets [default 0]\n"
	   " -g Stokes parameter to output to filterbank, from 0[I], 1[Q], 2[U], 3[V] [default 0]\n"
	   " -v minimum baseline length (E-W, in m) for input to beamformer [default 0]\n"
	   " -n do NOT normalize bf weights\n"
	   " -h print usage\n");
}


// MAIN

int main (int argc, char *argv[]) {

  // use cuda device 1
  printf("Using GPU 1\n");
  hipSetDevice(1);

  // command line arguments
  int arg = 0;
  char * finnam;
  finnam=(char *)malloc(sizeof(char)*100);
  char * foutnam;
  foutnam=(char *)malloc(sizeof(char)*100);
  int writing=0;
  int tint = 8;
  char * wnam;
  wnam=(char *)malloc(sizeof(char)*100);
  char * flagnam;
  flagnam=(char *)malloc(sizeof(char)*100);
  int weight=0, doflag=0;
  int beamn = -1;
  float fch1 = 1530.;
  char * filnam;
  filnam = (char *)malloc(sizeof(char)*100);
  int philwriting = 0;
  char * delnam;
  delnam = (char *)malloc(sizeof(char)*100);
  int delaying = 0;
  int averaging = 0;
  int dedispersing = 0;
  float dm = 0.;
  int OUTNT=NT;
  int OFFT=0;
  int unify=0;
  int stokes=0;
  float minBase=-1.;
  int donorm = 1;

  while ((arg=getopt(argc,argv,"i:o:t:w:f:c:b:p:d:m:s:q:g:v:nuah")) != -1)
    {
      switch (arg)
	{
	case 'i':
	  if (optarg)
	    {
	      strcpy(finnam,optarg);
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-i flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 'o':
	  if (optarg)
	    {
	      strcpy(foutnam,optarg);
	      writing=1;
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-o flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 'p':
	  if (optarg)
	    {
	      strcpy(filnam,optarg);
	      philwriting=1;
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-p flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 'd':
	  if (optarg)
	    {
	      strcpy(delnam,optarg);
	      delaying=1;
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-d flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 'v':
	  if (optarg)
	    {
	      minBase=atof(optarg);
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-v flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 'g':
	  if (optarg)
	    {
	      stokes=atoi(optarg);
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-g flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 'w':
	  if (optarg)
	    {
	      strcpy(wnam,optarg);
	      weight=1;
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-w flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 's':
	  if (optarg)
	    {
	      OUTNT = atoi(optarg);
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-s flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 'm':
	  if (optarg)
	    {
	      dm = atof(optarg);
	      dedispersing=1;
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-m flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 'q':
	  if (optarg)
	    {
	      OFFT = atoi(optarg);
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-q flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 'f':
	  if (optarg)
	    {
	      strcpy(flagnam,optarg);
	      doflag=1;
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-f flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 'c':
	  if (optarg)
	    {
	      fch1 = atof(optarg);
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-c flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 't':
	  if (optarg)
	    {
	      tint = atoi(optarg);
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-t flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
	case 'b':
	  if (optarg)
	    {
	      beamn = atoi(optarg);
	      break;
	    }
	  else
	    {
	      syslog(LOG_ERR,"-b flag requires argument");
	      usage();
	      return EXIT_FAILURE;
	    }
 	case 'a':
	  averaging=1;
	  break;
 	case 'n':
	  donorm=0;
	  break;
 	case 'u':
	  unify=1;
	  break;
	case 'h':
	  usage();
	  return EXIT_SUCCESS;
	}
    }

  if (writing) printf("Reading from %s, writing to %s\n",finnam,foutnam);
  else printf("Reading from %s, no visibilities written\n",finnam);
  if (philwriting) {
    printf("Will write coherent filterbank to %s\n",filnam);
    if (stokes<0 || stokes>3) {
      printf("Cannot form Stokes parameter %d\n",stokes);
      return EXIT_FAILURE;
    }
    printf("Using Stokes parameter %d\n",stokes);
    printf("Minimum baseline (m): %g\n",minBase);
  }
  printf("Integrating by %d ints - check that this is power of 2\n",tint);
  printf("Assuming fch1 %f MHz\n",fch1);
  if (weight) printf("Will weight voltages using %s\n",wnam);
  if (doflag) printf("Will flag antennas using %s\n",flagnam);
  if (beamn>=0 && beamn <=255) printf("Will rotate voltages to beam %d\n",beamn);
  else
    printf("Not rotating voltages with beamn %d\n",beamn);
  if (averaging) printf("Will average visibilities by 8x in frequency\n");
  if (delaying) printf("Will apply baseline delays from %s\n",delnam);
  if (dedispersing) printf("Will dedisperse to DM %f, adding delay to 1530MHz\n",dm);
  if (!donorm) printf("Will not normalize bf weights\n");

  // open input and output files
  FILE *fin, *fout, *flout;
  if (!(fin=fopen(finnam,"rb"))) 
    printf("could not open input file\n");
  if (writing) {
    if (!(fout=fopen(foutnam,"wb"))) 
      printf("could not open output file\n");
  }
  if (philwriting) {
    if (!(flout=fopen(filnam,"wb"))) 
      printf("could not open filterbank output file\n");
  }
  
  // read into memory and deal with dedispersion
  printf("initial memory allocation - please stay patient...\n");
  size_t asize = 2972712960;//NT*NANT*NCHAN*NPTR/((size_t)(2));
  size_t cpsize;
  char *indata = (char *)malloc(sizeof(char)*asize);
  char *d_alldata1, *d_alldata2;
  if (dedispersing)
    hipMalloc((void **)&d_alldata1, asize*sizeof(char));
  hipMalloc((void **)&d_alldata2, asize*sizeof(char));
  int *h_dms = (int *)malloc(sizeof(int)*NCHAN);
  int *d_dms;
  float myf;
  hipMalloc((void **)&d_dms, sizeof(int)*NCHAN);
  if (dedispersing) {
    for (int i=0;i<NCHAN;i++) {
      myf = (fch1 - i*250./8192.)*1e-3;
      h_dms[i] = (int)(round(4.15*dm*(pow(myf,-2.)-pow(1.53,-2.))/(0.065536)));
      //printf("DM delays %f (MHz) %d (samples)\n",myf*1e3,h_dms[i]);
    }
    hipMemcpy(d_dms, h_dms, NCHAN*sizeof(int), hipMemcpyHostToDevice);
  }
  printf("Reading from file...\n");
  fread(indata, sizeof(char), asize, fin);  
  printf("Reading onto GPU...\n");
  if (!dedispersing)
    hipMemcpy(d_alldata2, indata, asize*sizeof(char), hipMemcpyHostToDevice);
  else {
    hipMemcpy(d_alldata1, indata, asize*sizeof(char), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    printf("Dedispersing...\n");
    dedisperser<<<NT*NANT,NCHAN>>>(d_alldata1, d_alldata2, d_dms);
  }

  hipDeviceSynchronize();
  free(h_dms);
  free(indata);
  if (dedispersing)
    hipFree(d_alldata1);
  hipFree(d_dms);
  
  // allocate all memory

  // CPU
  float *outdata = (float *)malloc(sizeof(float)*NBASE*NCHAN*8);
  float *filout = (float *)malloc(sizeof(float)*NCHAN);
  int *h_a1 = (int *)malloc(sizeof(int)*NBASE);
  int *h_a2 = (int *)malloc(sizeof(int)*NBASE);
  // GPU
  char *d_indata;
  float *d_promoted, *d_corrout, *d_finalout, *d_avout;
  int *d_a1, *d_a2;
  float *d_filout;
  hipMalloc((void **)&d_indata, NANT*NCHAN*(NPTR/2)*sizeof(char));
  hipMalloc((void **)&d_promoted, NANT*NCHAN*NPTR*sizeof(float));
  hipMalloc((void **)&d_corrout, 2*NBASE*NCHAN*8*sizeof(float));
  hipMalloc((void **)&d_finalout, NBASE*NCHAN*8*sizeof(float));
  hipMalloc((void **)&d_avout, NBASE*(NCHAN/AV)*4*sizeof(float));
  hipMalloc((void **)&d_a1, NBASE*sizeof(int));
  hipMalloc((void **)&d_a2, NBASE*sizeof(int));
  hipMalloc((void **)&d_filout, NCHAN*sizeof(float));

  // load in delays
  float * h_delays = (float *)malloc(sizeof(float)*NBASE);
  float * d_delays;
  hipMalloc((void **)&d_delays, NBASE*sizeof(float));
  FILE *fdel;
  if (delaying) {
    if (!(fdel=fopen(delnam,"r"))) {
      printf("could not open delay file %s\n",delnam);
      return(1);
    }
    for (int i=0;i<NBASE;i++)
      fscanf(fdel,"%f\n",&h_delays[i]);
    fclose(fdel);
    hipMemcpy(d_delays,h_delays,NBASE*sizeof(float),hipMemcpyHostToDevice);
  }
  
  // load in weights and antpos
  float * antpos = (float *)malloc(sizeof(float)*64); // easting
  float * weights = (float *)malloc(sizeof(float)*64*48*2*2); // complex weights [ant, NW, pol, r/i]
  float * bfweights = (float *)malloc(sizeof(float)*64*48*2*2); // complex weights [ant, NW, pol, r/i]
  float * freqs = (float *)malloc(sizeof(float)*NCHAN); // freq
  float * d_freqs;
  hipMalloc((void **)&d_freqs, NCHAN*sizeof(float));
  for (int i=0;i<NCHAN;i++) freqs[i] = (fch1 - i*250./8192.)*1e6;
  hipMemcpy(d_freqs,freqs,NCHAN*sizeof(float),hipMemcpyHostToDevice);
  init_weights(wnam,antpos,weights,flagnam,weight,doflag,donorm);
  if (beamn>=0 && beamn<=255)
    calc_voltage_weights(antpos,weights,freqs,bfweights,beamn);
  float *d_weights;
  hipMalloc((void **)&d_weights, 64*48*2*2*sizeof(float));
  float *d_antpos;
  hipMalloc((void **)&d_antpos, 64*sizeof(float));
  hipMemcpy(d_antpos,antpos,64*sizeof(float),hipMemcpyHostToDevice);
  if (beamn>=0 && beamn<=255)
    hipMemcpy(d_weights,bfweights,64*48*2*2*sizeof(float),hipMemcpyHostToDevice);
  else
    hipMemcpy(d_weights,weights,64*48*2*2*sizeof(float),hipMemcpyHostToDevice);
  
  // set up a1 and a2
  int ctr=0;
  for (int i=0;i<63;i++) {
    for (int j=i;j<63;j++) {
      h_a1[ctr] = i;
      h_a2[ctr] = j;
      ctr++;
    }
  }
  hipMemcpy(d_a1,h_a1,NBASE*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_a2,h_a2,NBASE*sizeof(int),hipMemcpyHostToDevice);

  // loop over input

  printf("starting loop\n");
  
  int timi=0;
  ctr = 0;
  for (int bigI=OFFT;bigI<OUTNT+OFFT;bigI++) {

    // read data, send to GPU, promote
    cpsize = bigI*NANT*NCHAN*NPTR/2;
    hipMemcpy(d_indata, d_alldata2 + cpsize, (NANT*NCHAN*NPTR/2)*sizeof(char), hipMemcpyDeviceToDevice);
    promoter<<<NANT*NCHAN*NPTR/2/32,32>>>(d_indata, d_promoted);    
    //promoter<<<NANT*NCHAN*NPTR/2/32,32>>>(d_alldata2 + cpsize, d_promoted);
    if (unify) {
      for (int i=1;i<NANT;i++)
	hipMemcpy(d_promoted + i*NCHAN*NPTR, d_promoted, NCHAN*NPTR*sizeof(float), hipMemcpyDeviceToDevice);
    }
    
    // deal with time integration
    if (timi==0) {
      zeroer<<<2*NBASE*NCHAN*8/32,32>>>(d_corrout);
      zeroer<<<NBASE*NCHAN*8/32,32>>>(d_finalout);
    }
    // correlate
    correlator<<<NBASE*NCHAN/32,32>>>(d_promoted,d_corrout,d_a1,d_a2,(1./(1.*tint)),d_weights);
    timi+=2;

    // deal with time integration
    if (timi>=tint) {

      // don't add up
      if (tint==1) {
		
	if (writing) {
	  if (delaying)
	    delayer<<<NBASE*NCHAN*4/32, 32>>>(d_corrout, d_freqs, d_delays);
	  if (averaging) {
	    fscrunch<<<NBASE*NCHAN*4/AV/32, 32>>>(d_corrout, d_avout);
	    hipMemcpy(outdata, d_avout, NBASE*(NCHAN/AV)*4*sizeof(float), hipMemcpyDeviceToHost);
	    fwrite(outdata,sizeof(float),NBASE*(NCHAN/AV)*4,fout);
	  }
	  else {
	    hipMemcpy(outdata, d_corrout, NBASE*NCHAN*8*sizeof(float), hipMemcpyDeviceToHost);
	    fwrite(outdata,sizeof(float),NBASE*NCHAN*8,fout);
	  }
	}
	if (philwriting) {
	  reduce_corrs<<<NCHAN,512>>>(d_corrout, d_filout, 0.25, d_a1, d_a2, stokes, d_antpos, minBase);
	  hipMemcpy(filout, d_filout, NCHAN*sizeof(float), hipMemcpyDeviceToHost);
	  fwrite(filout,sizeof(float),NCHAN,flout);
	}
		
	if (writing) {
	  if (delaying)
	    delayer<<<NBASE*NCHAN*4/32, 32>>>(d_corrout + NBASE*NCHAN*8, d_freqs, d_delays);
	  if (averaging) {
	    fscrunch<<<NBASE*NCHAN*4/AV/32, 32>>>(d_corrout + NBASE*NCHAN*8, d_avout);
	    hipMemcpy(outdata, d_avout, NBASE*(NCHAN/AV)*4*sizeof(float), hipMemcpyDeviceToHost);
	    fwrite(outdata,sizeof(float),NBASE*(NCHAN/AV)*4,fout);
	  }
	  else {
	    hipMemcpy(outdata, d_corrout + NBASE*NCHAN*8, NBASE*NCHAN*8*sizeof(float), hipMemcpyDeviceToHost);
	    fwrite(outdata,sizeof(float),NBASE*NCHAN*8,fout);
	  }
	}
	if (philwriting) {
	  reduce_corrs<<<NCHAN,512>>>(d_corrout + NBASE*NCHAN*8, d_filout, 0.25, d_a1, d_a2, stokes, d_antpos, minBase);
	  hipMemcpy(filout, d_filout, NCHAN*sizeof(float), hipMemcpyDeviceToHost);
	  fwrite(filout,sizeof(float),NCHAN,flout);
	}
	
      }

      // add up
      else {

	adder<<<NBASE*NCHAN*8/32,32>>>(d_corrout,d_finalout);	
	if (writing) {
	  if (delaying)
	    delayer<<<NBASE*NCHAN*4/32, 32>>>(d_finalout, d_freqs, d_delays);
	  if (averaging) {
	    fscrunch<<<NBASE*NCHAN*4/AV/32, 32>>>(d_finalout, d_avout);
	    hipMemcpy(outdata, d_avout, NBASE*(NCHAN/AV)*4*sizeof(float), hipMemcpyDeviceToHost);
	    fwrite(outdata,sizeof(float),NBASE*(NCHAN/AV)*4,fout);
	  }
	  else {
	    hipMemcpy(outdata, d_finalout, NBASE*NCHAN*8*sizeof(float), hipMemcpyDeviceToHost);
	    fwrite(outdata,sizeof(float),NBASE*NCHAN*8,fout);
	  }
	}
	if (philwriting) {
	  reduce_corrs<<<NCHAN,512>>>(d_finalout, d_filout, 4., d_a1, d_a2, stokes, d_antpos, minBase);
	  hipMemcpy(filout, d_filout, NCHAN*sizeof(float), hipMemcpyDeviceToHost);	  
	  fwrite(filout,sizeof(float),NCHAN,flout);
	}

      }

      ctr++;
      //printf("done with integration %d of %d\n",ctr,NT*2/tint);
      timi = 0;
    }
      

  }

  fclose(fin);
  if (writing) fclose(fout);
  if (philwriting) fclose(flout);

  hipFree(d_alldata2);
  hipFree(d_indata);
  hipFree(d_corrout);
  hipFree(d_finalout);
  hipFree(d_promoted);
  hipFree(d_a1);
  hipFree(d_a2);
  hipFree(d_weights);
  hipFree(d_filout);
  hipFree(d_avout);
  hipFree(d_delays);
  hipFree(d_freqs);
  hipFree(d_antpos);
  free(filout);
  free(antpos);
  free(weights);
  free(freqs);
  free(wnam);
  free(flagnam);  
  free(outdata);
  free(h_a1);
  free(h_a2);
  free(finnam);
  free(foutnam);
  free(filnam);
  free(delnam);
  free(h_delays);

}
